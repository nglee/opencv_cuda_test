#include "hip/hip_runtime.h"


#include <stdio.h>
#include <thread>

__global__ void kernel(unsigned char* devPtr)
{
    devPtr[threadIdx.x + blockDim.x * blockIdx.x] = (unsigned char)sqrtf(threadIdx.x);
}

void func()
{
    while (1) {
        unsigned char* devPtr;
        hipError_t cudaStatus = hipMalloc(&devPtr, 1024 * 1024 * 1024);
        if (cudaStatus != hipSuccess) {
            fprintf(stdout, "(cudaMalloc) %s : %s\n", hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
            return;
        }
        else {
            fprintf(stdout, "(cudaMalloc) success\n");
        }

        kernel<<<40, 512>>>(devPtr);
        hipDeviceSynchronize();

        unsigned char* hPtr = (unsigned char*)malloc(1024 * 1024 * 1024);
        hipMemcpy(hPtr, devPtr, 1024 * 1024 * 1024, hipMemcpyDeviceToHost);

        cudaStatus = hipFree(devPtr);
        if (cudaStatus != hipSuccess) {
            fprintf(stdout, "(cudaFree) %s : %s\n", hipGetErrorName(cudaStatus), hipGetErrorString(cudaStatus));
            return;
        }
        else {
            fprintf(stdout, "(cudaFree) success\n");
        }

        free(hPtr);
    }
}

int main()
{
    std::thread t1(func);
    std::thread t2(func);
    std::thread t3(func);
    std::thread t4(func);

    while (1) {}
}