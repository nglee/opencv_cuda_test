#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <cstdlib>

template <typename T>
void _check(T result, char const* const func, char const* const file, int const line)
{
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), hipGetErrorName(result), func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
#define checkCudaErrors(val) _check((val), #val, __FILE__, __LINE__)

__global__
void matmul(const float* A, const float* B, float* C)
{
    
}

#define MATRIX_WIDTH (1ULL << 14)
#define MATRIX_HEIGHT (1ULL << 14)
#define MATRIX_SIZE (MATRIX_HEIGHT * MATRIX_WIDTH)
#define MALLOC_SIZE (sizeof(float) * MATRIX_SIZE)

int main()
{
    std::random_device rd;
    std::mt19937 e2(rd());
    std::uniform_real_distribution<float> dist(0, 1);

    // Allocate memory
    float* h_A = reinterpret_cast<float*>(std::malloc(MALLOC_SIZE));
    float* h_B = reinterpret_cast<float*>(std::malloc(MALLOC_SIZE));
    float* h_C = reinterpret_cast<float*>(std::malloc(MALLOC_SIZE));
    float *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc(&d_A, MALLOC_SIZE));
    checkCudaErrors(hipMalloc(&d_B, MALLOC_SIZE));
    checkCudaErrors(hipMalloc(&d_C, MALLOC_SIZE));

    // Init memory
    for (int idx = 0; idx < MATRIX_SIZE; idx++) {
        h_A[idx] = dist(e2);
        h_B[idx] = dist(e2);
    }

    // Copy host to device
    checkCudaErrors(hipMemcpy(d_A, h_A, MALLOC_SIZE, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, MALLOC_SIZE, hipMemcpyHostToDevice));

    dim3 gridSize(1024);
    dim3 blockSize(1024);

    matmul << <gridSize, blockSize >> > (d_A, d_B, d_C);

    // Copy device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, MALLOC_SIZE, hipMemcpyDeviceToHost));

    // Free memory
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_A));
    free(h_C);
    free(h_B);
    free(h_A);
}

