#include "hip/hip_runtime.h"
#include ""
#include "npp.h"
#include "nppi.h"

#include <omp.h>

#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <stdio.h>

using namespace cv;
using namespace cv::cuda;

    #define error_entry(entry)  { entry, #entry }

    struct ErrorEntry
    {
        int code;
        const char* str;
    };

    struct ErrorEntryComparer
    {
        int code;
        ErrorEntryComparer(int code_) : code(code_) {}
        bool operator()(const ErrorEntry& e) const { return e.code == code; }
    };

    const ErrorEntry npp_errors [] =
    {
    #if defined (_MSC_VER)
        error_entry( NPP_NOT_SUFFICIENT_COMPUTE_CAPABILITY ),
    #endif


        error_entry( NPP_INVALID_HOST_POINTER_ERROR ),
        error_entry( NPP_INVALID_DEVICE_POINTER_ERROR ),
        error_entry( NPP_LUT_PALETTE_BITSIZE_ERROR ),
        error_entry( NPP_ZC_MODE_NOT_SUPPORTED_ERROR ),
        error_entry( NPP_MEMFREE_ERROR ),
        error_entry( NPP_MEMSET_ERROR ),
        error_entry( NPP_QUALITY_INDEX_ERROR ),
        error_entry( NPP_HISTOGRAM_NUMBER_OF_LEVELS_ERROR ),
        error_entry( NPP_CHANNEL_ORDER_ERROR ),
        error_entry( NPP_ZERO_MASK_VALUE_ERROR ),
        error_entry( NPP_QUADRANGLE_ERROR ),
        error_entry( NPP_RECTANGLE_ERROR ),
        error_entry( NPP_COEFFICIENT_ERROR ),
        error_entry( NPP_NUMBER_OF_CHANNELS_ERROR ),
        error_entry( NPP_COI_ERROR ),
        error_entry( NPP_DIVISOR_ERROR ),
        error_entry( NPP_CHANNEL_ERROR ),
        error_entry( NPP_STRIDE_ERROR ),
        error_entry( NPP_ANCHOR_ERROR ),
        error_entry( NPP_MASK_SIZE_ERROR ),
        error_entry( NPP_MIRROR_FLIP_ERROR ),
        error_entry( NPP_MOMENT_00_ZERO_ERROR ),
        error_entry( NPP_THRESHOLD_NEGATIVE_LEVEL_ERROR ),
        error_entry( NPP_THRESHOLD_ERROR ),
        error_entry( NPP_CONTEXT_MATCH_ERROR ),
        error_entry( NPP_FFT_FLAG_ERROR ),
        error_entry( NPP_FFT_ORDER_ERROR ),
        error_entry( NPP_SCALE_RANGE_ERROR ),
        error_entry( NPP_DATA_TYPE_ERROR ),
        error_entry( NPP_OUT_OFF_RANGE_ERROR ),
        error_entry( NPP_DIVIDE_BY_ZERO_ERROR ),
        error_entry( NPP_MEMORY_ALLOCATION_ERR ),
        error_entry( NPP_RANGE_ERROR ),
        error_entry( NPP_BAD_ARGUMENT_ERROR ),
        error_entry( NPP_NO_MEMORY_ERROR ),
        error_entry( NPP_ERROR_RESERVED ),
        error_entry( NPP_NO_OPERATION_WARNING ),
        error_entry( NPP_DIVIDE_BY_ZERO_WARNING ),
        error_entry( NPP_WRONG_INTERSECTION_ROI_WARNING ),

        error_entry( NPP_NOT_SUPPORTED_MODE_ERROR ),
        error_entry( NPP_ROUND_MODE_NOT_SUPPORTED_ERROR ),
        error_entry( NPP_RESIZE_NO_OPERATION_ERROR ),
        error_entry( NPP_LUT_NUMBER_OF_LEVELS_ERROR ),
        error_entry( NPP_TEXTURE_BIND_ERROR ),
        error_entry( NPP_WRONG_INTERSECTION_ROI_ERROR ),
        error_entry( NPP_NOT_EVEN_STEP_ERROR ),
        error_entry( NPP_INTERPOLATION_ERROR ),
        error_entry( NPP_RESIZE_FACTOR_ERROR ),
        error_entry( NPP_HAAR_CLASSIFIER_PIXEL_MATCH_ERROR ),
        error_entry( NPP_MEMCPY_ERROR ),
        error_entry( NPP_ALIGNMENT_ERROR ),
        error_entry( NPP_STEP_ERROR ),
        error_entry( NPP_SIZE_ERROR ),
        error_entry( NPP_NULL_POINTER_ERROR ),
        error_entry( NPP_CUDA_KERNEL_EXECUTION_ERROR ),
        error_entry( NPP_NOT_IMPLEMENTED_ERROR ),
        error_entry( NPP_ERROR ),
        error_entry( NPP_NO_ERROR ),
        error_entry( NPP_SUCCESS ),
        error_entry( NPP_WRONG_INTERSECTION_QUAD_WARNING ),
        error_entry( NPP_MISALIGNED_DST_ROI_WARNING ),
        error_entry( NPP_AFFINE_QUAD_INCORRECT_WARNING ),
        error_entry( NPP_DOUBLE_SIZE_WARNING )
    };

    const size_t npp_error_num = sizeof(npp_errors) / sizeof(npp_errors[0]);

    cv::String getErrorString(int code, const ErrorEntry* errors, size_t n)
    {
        size_t idx = std::find_if(errors, errors + n, ErrorEntryComparer(code)) - errors;

        const char* msg = (idx != n) ? errors[idx].str : "Unknown error code";
        cv::String str = cv::format("%s [Code = %d]", msg, code);

        return str;
    }

String getNppErrorMessage(int code)
{
    return getErrorString(code, npp_errors, npp_error_num);
}

static inline void checkNppError(int code, const char* file, const int line, const char* func)
{
    if (code < 0)
        cv::error(cv::Error::GpuApiCallError, getNppErrorMessage(code), func, file, line);
}

#define nppSafeCall(expr)  checkNppError(expr, __FILE__, __LINE__, CV_Func)

int main()
{
    #pragma omp parallel num_threads(2)
    {
        // Source image : 512x512 1channel 8bit, all pixels set to 5
        GpuMat src(512, 512, CV_8UC1);
        src.setTo(Scalar(5));

        // Destination buffer where mean and stddev value is stored
        GpuMat dst(1, 2, CV_64FC1);

        // Create scratch buffer
        int bufSize;
        NppiSize sz;
        sz.width = src.cols;
        sz.height = src.rows;
        nppSafeCall( nppiMeanStdDevGetBufferHostSize_8u_C1R(sz, &bufSize) );
        GpuMat buf(1, bufSize, CV_8UC1);
        
        // Create stream
        hipStream_t stream;
        hipStreamCreate(&stream);

        // Set npp to use this stream
        nppSetStream(stream);

        nppSafeCall( nppiMean_StdDev_8u_C1R(src.ptr<Npp8u>(), static_cast<int>(src.step), sz, buf.ptr<Npp8u>(), dst.ptr<Npp64f>(), dst.ptr<Npp64f>() + 1) );

        // Wait until npp finish
        hipStreamSynchronize(stream);

        // Destroy stream
        hipStreamDestroy(stream);

        // Print output (expects mean = 5, stddev = 0)
        Mat h_dst;
        dst.download(h_dst);
        std::string out = "thread" + std::to_string(omp_get_thread_num()) + " : (mean)" + std::to_string(h_dst.at<Npp64f>(0, 0)) + " (stddev)" + std::to_string(h_dst.at<Npp64f>(0, 1)) + "\n";
        std::cout << out;
    }
}